#include "hip/hip_runtime.h"
#include "particle-data-structures.h"
#include "simulation-parameters.h"

#include "test-kernels.h"

#include <stdint.h>
#include <stdio.h>

__global__ void device_count_particles_in_grid_slots(gri_to_pl_map_t grid_to_particle_list_map,
                                              uint32_t *particles_per_grid_slot) {

    uint32_t num_grid_slots = (uint32_t)pow(EXP_SPACE_DIM / H, 3);

    for(size_t i = 0; i < num_grid_slots; i++) {
        uint16_t grid_count = 0;
        for(Particle *p = grid_to_particle_list_map[i]; p != NULL;
                p = p->next_particle) {
            grid_count++;
        }
        particles_per_grid_slot[i] = grid_count;
    }
}

void host_count_particles(gri_to_pl_map_t grid_to_particle_list_map,
                                              uint32_t *particles_per_grid_slot_forward,
                                              uint32_t *particles_per_grid_slot_backward) {

    uint32_t num_grid_slots = (uint32_t)pow(EXP_SPACE_DIM / H, 3);

    for(size_t i = 0; i < num_grid_slots; i++) {
        uint16_t grid_count_forward = 0;
        uint16_t grid_count_backward = 0;
        Particle *prev_particle = NULL;

        for(Particle *p = grid_to_particle_list_map[i]; p != NULL;
                p = p->next_particle) {
            prev_particle = p;
            grid_count_forward++;
        }
        particles_per_grid_slot_forward[i] = grid_count_forward;

        for(Particle *p = prev_particle; p != NULL;
                p = p->prev_particle) {
            grid_count_backward++;
        }
        particles_per_grid_slot_backward[i] = grid_count_backward;
    }
}

bool host_grid_consistency_check(gri_to_pl_map_t grid_to_particle_list_map) {

    uint32_t n_grid_slots = (uint32_t)pow(EXP_SPACE_DIM / H, 3);
    uint32_t *particles_per_grid_slot_forward;
    uint32_t *particles_per_grid_slot_backward;
    hipMallocManaged(&particles_per_grid_slot_forward, n_grid_slots * sizeof(uint32_t));
    hipMallocManaged(&particles_per_grid_slot_backward, n_grid_slots * sizeof(uint32_t));

    host_count_particles(grid_to_particle_list_map, particles_per_grid_slot_forward,
                                                    particles_per_grid_slot_backward);

    for(size_t i = 0; i < n_grid_slots; i++) {
        if(particles_per_grid_slot_forward[i] !=
           particles_per_grid_slot_backward[i]) {
            printf("Malformed dll at index %zu\n", i);
            return false;
        }

        if(particles_per_grid_slot_forward[i] != 0) {
            printf("Non-empty cow with %d particles\n",
                    particles_per_grid_slot_forward[i]);
        }
    }

    return true;
}

void output_curr_particle_to_grid_map(pi_to_gri_map_t curr_particle_to_grid_map) {

    for(size_t i = 0; i < N_PARTICLES; i++) {
        printf("particle idx, grid idx: %zu , %d\n", i, curr_particle_to_grid_map[i]);
    }
}
/*
__global__ void insert_particle_test(gri_to_pl_map_t grid_to_particle_list_map,
                                     pi_to_gri_map_t curr_particle_to_grid_map,
                                     pi_to_pa_map_t particle_idx_to_addr_map,
                                     grid_mutex_set_t mutex_set) {

    uint32_t particle_idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t grid_idx = particle_idx % 8;

    Particle *new_particle = particle_idx_to_addr_map[particle_idx];

    device_insert_into_grid(grid_to_particle_list_map,
                            grid_idx,
                            curr_particle_to_grid_map,
                            particle_idx,
                            new_particle,
                            mutex_set);
}


__global__ void delete_particles_test(gri_to_pl_map_t grid_to_particle_list_map,
                                      pi_to_gri_map_t curr_particle_to_grid_map,
                                      pi_to_pa_map_t particle_idx_to_addr_map,
                                      grid_mutex_set_t mutex_set) {

    uint32_t particle_idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t grid_idx = curr_particle_to_grid_map[particle_idx];

    Particle *del_particle = particle_idx_to_addr_map[particle_idx];

    device_remove_from_grid(grid_to_particle_list_map,
                            grid_idx,
                            del_particle,
                            mutex_set);
}
*/


