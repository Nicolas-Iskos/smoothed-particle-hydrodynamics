#include "hip/hip_runtime.h"
#include "particle-data-structures.h"
#include "simulation-parameters.h"

#include <stdint.h>


pi_to_gri_map_t gen_particle_to_grid_map() {
    pi_to_gri_map_t particle_to_grid_map;

    hipMallocManaged(&particle_to_grid_map,
                      N_PARTICLES * sizeof(uint32_t));

    return particle_to_grid_map;
}


gri_to_pl_map_t gen_grid_to_particle_list_map() {
    gri_to_pl_map_t grid_to_particle_list_map;
    uint32_t n_grid_spaces;

    n_grid_spaces = (uint32_t)pow(EXP_SPACE_DIM / H, 3);

    hipMallocManaged(&grid_to_particle_list_map,
            n_grid_spaces * sizeof(Particle*));

    /* ensure that each doubly linked list is NULL-terminated */
    for(size_t i = 0; i < n_grid_spaces; i++) {
        grid_to_particle_list_map[i] = NULL;
    }

    return grid_to_particle_list_map;
}


pi_to_pa_map_t gen_particle_idx_to_addr_map() {
    pi_to_pa_map_t particle_idx_to_addr_map;

    hipMallocManaged(&particle_idx_to_addr_map,
            N_PARTICLES * sizeof(Particle*));

    return particle_idx_to_addr_map;
}

/*
 * The dam break initialization function creates a cubic block
 * of particles arranged into a simple cubic lattice centered in
 * the experiment space
 *
 * the experiment space exists in a right handed cartesian
 * coordinate system
 * */
void initialize_dam_break(gri_to_pl_map_t grid_to_particle_list_map,
                          pi_to_gri_map_t last_particle_to_grid_map,
                          pi_to_gri_map_t curr_particle_to_grid_map,
                          pi_to_pa_map_t particle_idx_to_addr_map) {

    uint32_t n_particles_per_dim;
    uint32_t n_particles_per_dim_pow2;
    float cubic_block_rad;
    float particle_spacing;
    float space_center[3];
    float init_particle_pos[3];
    float particle_pos[3];
    uint32_t grid_idx;
    uint32_t particle_idx;
    Particle *new_particle;



    n_particles_per_dim = (uint32_t)cbrt((float)N_PARTICLES);
    n_particles_per_dim_pow2 = (uint32_t)pow(n_particles_per_dim, 2);

    cubic_block_rad = n_particles_per_dim * R_PARTICLE;
    particle_spacing = 2 * R_PARTICLE;

    space_center[0] = (float)EXP_SPACE_DIM / 2;
    space_center[1] = (float)EXP_SPACE_DIM / 2;
    space_center[2] = (float)EXP_SPACE_DIM / 2;

    init_particle_pos[0] = space_center[0] + cubic_block_rad - R_PARTICLE;
    init_particle_pos[1] = space_center[1] - cubic_block_rad + R_PARTICLE;
    init_particle_pos[2] = space_center[2] + cubic_block_rad - R_PARTICLE;

    /*
     * Arrange each particle into its correct grid slot for the
     * simple cubic lattice arrangement
     *
     * Looking down the x-axis towards the origin, we build each
     * slice of the lattice perpendicular to the x-axis
     * from top left to bottom right, proceeding
     * along horizontal rows. The slices of the lattice are built
     * starting at high x-values, going to low x-values.
     * */
    for(particle_idx = 0; particle_idx < N_PARTICLES; particle_idx++) {
        /* compute the position of the particle to be created */
        particle_pos[0] =
            init_particle_pos[0] - particle_spacing *
            (particle_idx / n_particles_per_dim_pow2);
        particle_pos[1] =
            init_particle_pos[1] + particle_spacing *
            (particle_idx % n_particles_per_dim);
        particle_pos[2] =
            init_particle_pos[2] - particle_spacing *
            ((particle_idx % n_particles_per_dim_pow2) / n_particles_per_dim);

        /* initialize the new particle */
        new_particle = new Particle;

        new_particle->position[0] = particle_pos[0];
        new_particle->position[1] = particle_pos[1];
        new_particle->position[2] = particle_pos[2];

        new_particle->velocity[0] = 0;
        new_particle->velocity[1] = 0;
        new_particle->velocity[2] = 0;

        new_particle->force[0] = 0;
        new_particle->force[1] = 0;
        new_particle->force[2] = 0;

        new_particle->density = 0;
        new_particle->pressure = 0;
        new_particle->internal_energy = 0;

        /* record the address of the new particle */
        particle_idx_to_addr_map[particle_idx] = new_particle;

        /* record the grid index of each particle */
        grid_idx = calculate_grid_idx(new_particle->position);
        curr_particle_to_grid_map[particle_idx] = grid_idx;

        /*
         * insert the new particle into the correct grid space and
         * record the grid space of the new particle
         * */
        host_insert_into_grid(grid_to_particle_list_map,
                              grid_idx,
                              new_particle);
    }
}


void host_insert_into_grid(gri_to_pl_map_t grid_to_particle_list_map,
                           uint32_t grid_idx,
                           Particle *new_particle) {

    Particle *first_particle_in_grid_slot;

    first_particle_in_grid_slot = grid_to_particle_list_map[grid_idx];

    /* add particle to the correct grid space doubly linked list */
    if(first_particle_in_grid_slot == NULL) {
        new_particle->prev_particle = NULL;
        new_particle->next_particle = NULL;
        grid_to_particle_list_map[grid_idx] = new_particle;
    }
    else {
        first_particle_in_grid_slot->prev_particle = new_particle;
        new_particle->next_particle = first_particle_in_grid_slot;
        new_particle->prev_particle = NULL;
        grid_to_particle_list_map[grid_idx] = new_particle;
    }
}


__global__ void update_particle_to_grid_map(
                                pi_to_pa_map_t particle_idx_to_addr_map,
                                pi_to_gri_map_t last_particle_to_grid_map,
                                pi_to_gri_map_t curr_particle_to_grid_map) {
    uint32_t particle_idx;
    uint32_t pre_update_grid_idx;
    uint32_t updated_grid_idx;
    Particle *particle;

    particle_idx = blockDim.x * blockIdx.x + threadIdx.x;
    particle = particle_idx_to_addr_map[particle_idx];
    pre_update_grid_idx = curr_particle_to_grid_map[particle_idx];
    updated_grid_idx = calculate_grid_idx(particle->position);

    /* set the pre-updated grid_idx in the last particle to grid map */
    last_particle_to_grid_map[particle_idx] = pre_update_grid_idx;

    /* set the updated grid idx into the current particle to grid map */
    curr_particle_to_grid_map[particle_idx] = updated_grid_idx;
}


__global__ void perform_removals_from_grid(
                                gri_to_pl_map_t grid_to_particle_list_map,
                                pi_to_gri_map_t last_particle_to_grid_map,
                                pi_to_gri_map_t curr_particle_to_grid_map,
                                pi_to_pa_map_t particle_idx_to_addr_map) {
    uint32_t grid_idx;
    Particle *del_particle;
    Particle *del_prev_particle;
    Particle *del_next_particle;

    grid_idx = blockDim.x * blockIdx.x + threadIdx.x;

    for(uint32_t particle_idx = 0; particle_idx < N_PARTICLES; particle_idx++){
        if((last_particle_to_grid_map[particle_idx] == grid_idx) &&
           (curr_particle_to_grid_map[particle_idx] != grid_idx)) {

            del_particle = particle_idx_to_addr_map[particle_idx];
            del_prev_particle = del_particle->prev_particle;
            del_next_particle = del_particle->next_particle;

            if(del_prev_particle == NULL && del_next_particle == NULL) {
                grid_to_particle_list_map[grid_idx] = NULL;
            }
            else if(del_prev_particle == NULL) {
                grid_to_particle_list_map[grid_idx] = del_next_particle;
                del_next_particle->prev_particle = NULL;
            }
            else if(del_next_particle == NULL) {
                del_prev_particle->next_particle = NULL;
            }
            else {
                del_prev_particle->next_particle = del_next_particle;
                del_next_particle->prev_particle = del_prev_particle;
            }
        }
    }
}


__global__ void perform_additions_to_grid(
                                gri_to_pl_map_t grid_to_particle_list_map,
                                pi_to_gri_map_t last_particle_to_grid_map,
                                pi_to_gri_map_t curr_particle_to_grid_map,
                                pi_to_pa_map_t particle_idx_to_addr_map) {
    uint32_t grid_idx;
    Particle *particle;
    Particle *first_particle_in_grid_slot;

    grid_idx = blockDim.x * blockIdx.x + threadIdx.x;

    for(uint32_t particle_idx = 0; particle_idx < N_PARTICLES; particle_idx++){
        if((last_particle_to_grid_map[particle_idx] != grid_idx) &&
           (curr_particle_to_grid_map[particle_idx] == grid_idx)) {

            particle = particle_idx_to_addr_map[particle_idx];
            first_particle_in_grid_slot = grid_to_particle_list_map[grid_idx];

            /* add particle to the correct grid space doubly linked list */
            if(first_particle_in_grid_slot == NULL) {
                particle->prev_particle = NULL;
                particle->next_particle = NULL;
                grid_to_particle_list_map[grid_idx] = particle;
            }
            else {
                first_particle_in_grid_slot->prev_particle = particle;
                particle->next_particle = first_particle_in_grid_slot;
                particle->prev_particle = NULL;
                grid_to_particle_list_map[grid_idx] = particle;
            }
        }
    }
}


__host__ __device__ uint32_t calculate_grid_idx(float position[]) {
    uint32_t grid_space_layer;
    uint32_t grid_space_col;
    uint32_t grid_space_row;
    uint32_t grid_idx;
    constexpr uint32_t n_grid_spaces_per_dim = (uint32_t)(EXP_SPACE_DIM / H);
    constexpr uint32_t n_grid_spaces_per_dim_pow2 =
                       (uint32_t)((EXP_SPACE_DIM * EXP_SPACE_DIM) / (H * H));

    grid_space_layer = (uint16_t)((EXP_SPACE_DIM - position[0]) / H);
    grid_space_col = (uint16_t)(position[1] / H);
    grid_space_row = (uint16_t)((EXP_SPACE_DIM - position[2]) / H);

    grid_idx = grid_space_col +
               grid_space_row * n_grid_spaces_per_dim +
               grid_space_layer * n_grid_spaces_per_dim_pow2;

    return grid_idx;
}
