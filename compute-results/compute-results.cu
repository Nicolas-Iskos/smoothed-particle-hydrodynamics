#include "hip/hip_runtime.h"
#include "../simulation-parameters.h"

#include "particle-data-structures.h"
#include "calculate-field.h"
#include "integrate.h"
#include <fstream>
#include <sstream>
#include <iostream>

int main(int argc, char **argv) {

    float n_seconds_run_time;
    uint16_t n_iterations;
    std::string saveDirectory;
    Particle *particle;




    if(argc != 3) {
        std::cout << "Incorrect number of input arguments" << std::endl;
        return -1;
    }

    /* Determine how many iterations the simulation should complete */
    n_seconds_run_time = std::stof(argv[1]);
    n_iterations = (uint16_t)(n_seconds_run_time / DT);

    saveDirectory = argv[2];
    /* initialize simulation results output file */
    std::ofstream output(saveDirectory.append("/simulation-results.csv"),
                          std::ios::trunc);



    /* initialize particle data structures with dam break configuration */
    constexpr uint32_t n_grid_spaces = (EXP_SPACE_DIM * EXP_SPACE_DIM * EXP_SPACE_DIM) /
                                       (H * H * H);

    gri_to_pl_map_t grid_to_particle_list_map = gen_grid_to_particle_list_map();

    pi_to_gri_map_t last_particle_to_grid_map = gen_particle_to_grid_map();
    pi_to_gri_map_t curr_particle_to_grid_map = gen_particle_to_grid_map();

    pi_to_pa_map_t particle_idx_to_addr_map = gen_particle_idx_to_addr_map();

    initialize_dam_break(grid_to_particle_list_map, last_particle_to_grid_map,
                         curr_particle_to_grid_map, particle_idx_to_addr_map);




    /* perform the simulation */
    for(uint16_t i = 0; i < n_iterations; i++) {
        /* compute the forces acting on each particle using SPH techniques */
        calculate_density<<<N_PARTICLES / PARTICLES_PER_BLOCK,
                            PARTICLES_PER_BLOCK>>>(grid_to_particle_list_map,
                                                   curr_particle_to_grid_map,
                                                   particle_idx_to_addr_map);
        hipDeviceSynchronize();

        calculate_pressure<<<N_PARTICLES / PARTICLES_PER_BLOCK,
                             PARTICLES_PER_BLOCK>>>(particle_idx_to_addr_map);
        hipDeviceSynchronize();

        calculate_net_force<<<N_PARTICLES / PARTICLES_PER_BLOCK,
                              PARTICLES_PER_BLOCK>>>(grid_to_particle_list_map,
                                                     curr_particle_to_grid_map,
                                                     particle_idx_to_addr_map);
        hipDeviceSynchronize();



        /* integrate the position and velocity of the particles based on the
         * recently-computed net force acting on each particle
         * */
        euler_integrate<<<N_PARTICLES / PARTICLES_PER_BLOCK,
                          PARTICLES_PER_BLOCK>>>(particle_idx_to_addr_map);
        hipDeviceSynchronize();

        /* ensure that no particle passes into the outer layer of grid spaces
         * in the experimental space, or out of the experimental space
         * entirely
         * */
        enforce_boundary_conditions<<<N_PARTICLES / PARTICLES_PER_BLOCK,
                                      PARTICLES_PER_BLOCK>>>(particle_idx_to_addr_map);
        hipDeviceSynchronize();



        /* update the particle grid in 3 steps:
         *
         * 1. Keep track of what grid spaces the particles were last in, but also
         *    recompute what their new grid spaces should be and record these
         *    grid spaces as well
         *
         * 2. For each grid space, remove all particles that have left that grid
         *    space.
         *
         * 3. For each grid space, add all particles that have entered that
         *    grid space.
         * */
        update_particle_to_grid_map<<<n_grid_spaces / GRID_SPACES_PER_BLOCK,
                                      GRID_SPACES_PER_BLOCK>>>(last_particle_to_grid_map,
                                                               curr_particle_to_grid_map,
                                                               particle_idx_to_addr_map);
        hipDeviceSynchronize();

        perform_removals_from_grid<<<n_grid_spaces / GRID_SPACES_PER_BLOCK,
                                     GRID_SPACES_PER_BLOCK>>>(grid_to_particle_list_map,
                                                              last_particle_to_grid_map,
                                                              curr_particle_to_grid_map,
                                                              particle_idx_to_addr_map);
        hipDeviceSynchronize();

        perform_additions_to_grid<<<n_grid_spaces / GRID_SPACES_PER_BLOCK,
                                     GRID_SPACES_PER_BLOCK>>>(grid_to_particle_list_map,
                                                              last_particle_to_grid_map,
                                                              curr_particle_to_grid_map,
                                                              particle_idx_to_addr_map);
        hipDeviceSynchronize();



        /* this is a string stream holding the position of every particle as
         * a series of tuples laid out in a row
         * */
        std::stringstream particle_positions_stream;
        /* fill the newly-created stream with the position of each particle */
        for(uint32_t particle_idx = 0; particle_idx < N_PARTICLES; particle_idx++) {
            particle = particle_idx_to_addr_map[particle_idx];
            std::stringstream position_stream;

            position_stream << round(particle->position[0] * 1e4) / 1e4;
            position_stream << " ";
            position_stream << round(particle->position[1] * 1e4) / 1e4;
            position_stream << " ";
            position_stream << round(particle->position[2] * 1e4) / 1e4;
            position_stream << "    ";

            particle_positions_stream << position_stream.str();
        }

        /* add the stream as a line of the output file */
        output << particle_positions_stream.str() << std::endl;
    }
    return 0;
}
