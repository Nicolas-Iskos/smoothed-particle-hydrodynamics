#include "hip/hip_runtime.h"
/*
 * When compiled and run, this file will run the simulation and export its
 * results to a file named "simulation-results.csv."
 * */

#include "../simulation-parameters.h"

#include "particle-data-structures.h"
#include "calculate-field.h"
#include "integrate.h"
#include <fstream>
#include <sstream>
#include <iostream>

#include <cmath>



/*
 * main runs the simulation by first initializing the variables necessary
 * to run the simulation, and then running the simulation by updating
 * the parameters of each particle after successive time steps of length
 * DT.
 *
 * Inputs:
 * argc - the number of command line arguments (should be 3)
 * agrv - an array of the command line arguments: the first argument is
 *        the name of the executable, the second is the number of seconds
 *        for which you would like the simulation to run, and the third
 *        is the file path of the directory to which you would like to
 *        export the simulation results.
 *
 * Outputs: None
 * */
int main(int argc, char **argv) {

    float n_seconds_run_time;
    uint16_t n_iterations;
    std::string save_path;
    std::string full_save_path;
    Particle *particle;




    if(argc != 3) {
        std::cout << "Incorrect number of input arguments" << std::endl;
        return -1;
    }

    /* Determine how many iterations the simulation should complete */
    n_seconds_run_time = std::stof(argv[1]);
    n_iterations = (uint16_t)(n_seconds_run_time / DT);

    save_path = argv[2];
    full_save_path = save_path.append("/simulation-results.csv");
    /* initialize simulation results output file */
    std::ofstream output(full_save_path, std::ios::trunc);



    /* initialize particle data structures with dam break configuration */
    constexpr uint32_t n_grid_spaces = (EXP_SPACE_DIM * EXP_SPACE_DIM * EXP_SPACE_DIM) /
                                       (H * H * H);

    gri_to_pl_map_t grid_to_particle_list_map = gen_grid_to_particle_list_map();

    pi_to_gri_map_t last_particle_to_grid_map = gen_particle_to_grid_map();
    pi_to_gri_map_t curr_particle_to_grid_map = gen_particle_to_grid_map();

    pi_to_pa_map_t particle_idx_to_addr_map = gen_particle_idx_to_addr_map();

    initialize_dam_break(grid_to_particle_list_map, last_particle_to_grid_map,
                         curr_particle_to_grid_map, particle_idx_to_addr_map);




    /* run the simulation */
    for(uint16_t i = 0; i < n_iterations; i++) {
        /* compute the forces acting on each particle using SPH techniques */
        calculate_density<<<N_PARTICLES / PARTICLES_PER_BLOCK,
                            PARTICLES_PER_BLOCK>>>(grid_to_particle_list_map,
                                                   curr_particle_to_grid_map,
                                                   particle_idx_to_addr_map);
        hipDeviceSynchronize();


        calculate_pressure<<<N_PARTICLES / PARTICLES_PER_BLOCK,
                             PARTICLES_PER_BLOCK>>>(particle_idx_to_addr_map);
        hipDeviceSynchronize();


        calculate_net_force<<<N_PARTICLES / PARTICLES_PER_BLOCK,
                              PARTICLES_PER_BLOCK>>>(grid_to_particle_list_map,
                                                     curr_particle_to_grid_map,
                                                     particle_idx_to_addr_map);
        hipDeviceSynchronize();


        /* integrate the position and velocity of the particles based on the
         * recently-computed net force acting on each particle
         * */
        euler_integrate<<<N_PARTICLES / PARTICLES_PER_BLOCK,
                          PARTICLES_PER_BLOCK>>>(particle_idx_to_addr_map);
        hipDeviceSynchronize();


        /* ensure that no particle passes into the outer layer of grid spaces
         * in the experimental space, or out of the experimental space
         * entirely
         * */
        enforce_boundary_conditions<<<N_PARTICLES / PARTICLES_PER_BLOCK,
                                      PARTICLES_PER_BLOCK>>>(particle_idx_to_addr_map);
        hipDeviceSynchronize();



        /* update the particle grid in 3 steps:
         *
         * 1. Keep track of what grid spaces the particles were last in, but also
         *    recompute what their new grid spaces should be and record these
         *    grid spaces as well
         *
         * 2. For each grid space, remove all particles that have left that grid
         *    space.
         *
         * 3. For each grid space, add all particles that have entered that
         *    grid space.
         * */
        update_particle_to_grid_map<<<n_grid_spaces / GRID_SPACES_PER_BLOCK,
                                      GRID_SPACES_PER_BLOCK>>>(last_particle_to_grid_map,
                                                               curr_particle_to_grid_map,
                                                               particle_idx_to_addr_map);
        hipDeviceSynchronize();


        perform_removals_from_grid<<<n_grid_spaces / GRID_SPACES_PER_BLOCK,
                                     GRID_SPACES_PER_BLOCK>>>(grid_to_particle_list_map,
                                                              last_particle_to_grid_map,
                                                              curr_particle_to_grid_map,
                                                              particle_idx_to_addr_map);
        hipDeviceSynchronize();


        perform_additions_to_grid<<<n_grid_spaces / GRID_SPACES_PER_BLOCK,
                                     GRID_SPACES_PER_BLOCK>>>(grid_to_particle_list_map,
                                                              last_particle_to_grid_map,
                                                              curr_particle_to_grid_map,
                                                              particle_idx_to_addr_map);
        hipDeviceSynchronize();


        /* create a stream to hold the positions of every particle at the current
         * time step
         * */
        std::stringstream particle_positions_stream;
        /* fill the newly-created stream with the position of each particle */
        for(uint32_t particle_idx = 0; particle_idx < N_PARTICLES; particle_idx++) {
            particle = particle_idx_to_addr_map[particle_idx];
            std::stringstream position_stream;

            position_stream << round(particle->position[0] * 1e4) / 1e4;
            position_stream << " ";
            position_stream << round(particle->position[1] * 1e4) / 1e4;
            position_stream << " ";
            position_stream << round(particle->position[2] * 1e4) / 1e4;
            position_stream << "    ";

            particle_positions_stream << position_stream.str();
        }

        /* add the stream as a line of the output file */
        output << particle_positions_stream.str() << std::endl;
    }
}
